
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define DATA_WIDTH 16384
#define COVN_WIDTH 2
__constant__ int conv[COVN_WIDTH*COVN_WIDTH];//Convolution kernel placed in the constant region
__global__ void img2col_cau(int* in,int * out)//kernel-function
{
int bidx=blockIdx.x;
int thdx=threadIdx.x;
int bck_size=blockDim.x;
int sum=0;
while(thdx<(DATA_WIDTH-COVN_WIDTH+1))//conv_calu
{
    for(int i=0;i<COVN_WIDTH*COVN_WIDTH;i++)
{sum+=in[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)*4+i]*conv[i];}
out[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)]=sum;
sum=0;
thdx+=bck_size;
}
__syncthreads();

}
int main()
{
    int N=DATA_WIDTH*DATA_WIDTH;
    int M=COVN_WIDTH*COVN_WIDTH;
    int *h_data_in,*h_data_use,*h_data_out,*h_conv_in,*d_data_use,*d_data_out,*d_conv_use;
    h_data_in=(int*)malloc(sizeof(int)*N);
    h_conv_in=(int*)malloc(sizeof(int)*M);
    h_data_use=(int*)malloc(sizeof(int)*M*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    h_data_out=(int*)malloc(sizeof(int)*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    for(int i=0;i<DATA_WIDTH;i++)
    {
        for(int j=0;j<DATA_WIDTH;j++)
        {
            h_data_in[i*DATA_WIDTH+j]=2;
        }
    }
    for(int i=0;i<COVN_WIDTH;i++)
    {
        for(int j=0;j<COVN_WIDTH;j++)
        {
            h_conv_in[i*COVN_WIDTH+j]=3;
        }
    }
    // cudaEvent_t time_start,time_stop;
    // cudaEventCreate(&time_start);
    // cudaEventCreate(&time_stop);
    // cudaEventRecord(time_start,0);
    // cudaEventSynchronize(time_start);
    hipMalloc((void**)&d_conv_use,sizeof(int)*M);
    hipMalloc((void**)&d_data_use,sizeof(int)*M*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    hipMalloc((void**)&d_data_out,sizeof(int)*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    for(int i=0;i<(DATA_WIDTH-COVN_WIDTH+1);i++)//data_preprocess
    {
        for(int j=0;j<(DATA_WIDTH-COVN_WIDTH+1);j++)
        {
            h_data_use[(i*(DATA_WIDTH-COVN_WIDTH+1)+j)*4]=h_data_in[(i*(DATA_WIDTH-COVN_WIDTH+1)+j)];
            h_data_use[(i*(DATA_WIDTH-COVN_WIDTH+1)+j)*4+1]=h_data_in[(i*(DATA_WIDTH-COVN_WIDTH+1)+j+1)];
            h_data_use[(i*(DATA_WIDTH-COVN_WIDTH+1)+j)*4+2]=h_data_in[((i+1)*(DATA_WIDTH-COVN_WIDTH+1)+j)];
            h_data_use[(i*(DATA_WIDTH-COVN_WIDTH+1)+j)*4+3]=h_data_in[((i+1)*(DATA_WIDTH-COVN_WIDTH+1)+j+1)];
        }
    }
    hipMemcpy(d_data_use,h_data_use,sizeof(int)*M*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1),hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(conv),h_conv_in,sizeof(int)*M);


    img2col_cau<<<(DATA_WIDTH-COVN_WIDTH+1),64>>>(d_data_use,d_data_out);
    hipDeviceSynchronize();

    // cudaEventRecord(time_stop,0);
    // cudaEventSynchronize(time_stop);
    // float elapsedTime;
    // cudaEventElapsedTime(&elapsedTime,time_start,time_stop);
    // printf("runtime:%f(ms)\n",elapsedTime);

    hipMemcpy(h_data_out,d_data_out,sizeof(int)*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1),hipMemcpyDeviceToHost);
// for(int i=0;i<(DATA_WIDTH-COVN_WIDTH+1);i++)
// {
//     for(int j=0;j<(DATA_WIDTH-COVN_WIDTH+1);j++)
//     {
//         printf(" %d ",h_data_out[i*(DATA_WIDTH-COVN_WIDTH+1)+j]);
//     }
//     printf("\n");
// }

    hipFree(d_conv_use);
    hipFree(d_data_out);
    hipFree(d_data_use);
    free(h_data_in);
    free(h_data_use);
    free(h_conv_in);
    free(h_data_out);

}
