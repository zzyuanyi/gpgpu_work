
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define DATA_WIDTH 16384
#define COVN_WIDTH 2
__constant__ int conv[COVN_WIDTH*COVN_WIDTH];//Convolution kernel placed in the constant region
__global__ void img2col_cau(int* in,int * out,int*d_data_use)//kernel function
{
int bidx=blockIdx.x;
int thdx=threadIdx.x;
int bck_size=blockDim.x;
int sum=0;
while(thdx<(DATA_WIDTH-COVN_WIDTH+1))//data preprocrss
{
            d_data_use[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)*4]=in[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)];
            d_data_use[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)*4+1]=in[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx+1)];
            d_data_use[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)*4+2]=in[((bidx+1)*(DATA_WIDTH-COVN_WIDTH+1)+thdx)];
            d_data_use[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)*4+3]=in[((bidx+1)*(DATA_WIDTH-COVN_WIDTH+1)+thdx+1)];
            thdx+=bck_size;
}

thdx=threadIdx.x;

while(thdx<(DATA_WIDTH-COVN_WIDTH+1))//conv_calu
{
    for(int i=0;i<COVN_WIDTH*COVN_WIDTH;i++)
{sum+=d_data_use[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)*4+i]*conv[i];}
out[(bidx*(DATA_WIDTH-COVN_WIDTH+1)+thdx)]=sum;
sum=0;
thdx+=bck_size;
}
__syncthreads();

}
int main()
{
    int N=DATA_WIDTH*DATA_WIDTH;
    int M=COVN_WIDTH*COVN_WIDTH;
    int *h_data_in,*h_data_use,*h_data_out,*h_conv_in,*d_data_use,*d_data_out,*d_data_in;//No need to perform the flatten operation again because the convolution kernel is too small
    h_data_in=(int*)malloc(sizeof(int)*N);
    h_conv_in=(int*)malloc(sizeof(int)*M);
    h_data_use=(int*)malloc(sizeof(int)*M*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    h_data_out=(int*)malloc(sizeof(int)*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    for(int i=0;i<DATA_WIDTH;i++)
    {
        for(int j=0;j<DATA_WIDTH;j++)
        {
            h_data_in[i*DATA_WIDTH+j]=2;
        }
    }
    for(int i=0;i<COVN_WIDTH;i++)
    {
        for(int j=0;j<COVN_WIDTH;j++)
        {
            h_conv_in[i*COVN_WIDTH+j]=3;
        }
    }
    // cudaEvent_t time_start,time_stop;
    // cudaEventCreate(&time_start);
    // cudaEventCreate(&time_stop);
    // cudaEventRecord(time_start,0);
    // cudaEventSynchronize(time_start);
    hipMalloc((void**)&d_data_use,sizeof(int)*M*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));
    hipMalloc((void**)&d_data_in,sizeof(int)*N);
    hipMalloc((void**)&d_data_out,sizeof(int)*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1));

    hipMemcpy(d_data_in,h_data_in,sizeof(int)*N,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(conv),h_conv_in,sizeof(int)*M);


    img2col_cau<<<(DATA_WIDTH-COVN_WIDTH+1),64>>>(d_data_in,d_data_out,d_data_use);
    hipDeviceSynchronize();

    // cudaEventRecord(time_stop,0);
    // cudaEventSynchronize(time_stop);
    // float elapsedTime;
    // cudaEventElapsedTime(&elapsedTime,time_start,time_stop);
    // printf("runtime:%f(ms)\n",elapsedTime);

    hipMemcpy(h_data_out,d_data_out,sizeof(int)*(DATA_WIDTH-COVN_WIDTH+1)*(DATA_WIDTH-COVN_WIDTH+1),hipMemcpyDeviceToHost);
// for(int i=0;i<(DATA_WIDTH-COVN_WIDTH+1);i++)
// {
//     for(int j=0;j<(DATA_WIDTH-COVN_WIDTH+1);j++)
//     {
//         printf(" %d ",h_data_out[i*(DATA_WIDTH-COVN_WIDTH+1)+j]);
//     }
//     printf("\n");
// }
// Annotation segments for timing and test results module

    hipFree(d_data_out);
    hipFree(d_data_use);
    hipFree(d_data_in);
    free(h_data_in);
    free(h_data_use);
    free(h_conv_in);
    free(h_data_out);

}
